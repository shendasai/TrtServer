#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cassert>
#include <cstring>
#include <vector>

#include "NvInfer.h"
#include "common.h"
#include "embLayerNormPlugin.h"
#include "logger.h"
#include "pluginKernels.h"
#include "pluginUtil.h"

using namespace nvinfer1;
using bert::operator+;

namespace bert
{

namespace test
{

template <typename T, unsigned TPB>
__global__ void embLayerNormKernel(int ld, const int* inputIds, const int* tokenIds, const float* beta,
    const float* gamma, const float* wordEmb, const float* posEmb, const float* tokEmb, T* output)
{

    hipcub::Sum pairSum;
    // 1. lookup word and token of the block
    // blockIdx.x = position in the sequence
    // blockIdx.y = batch
    // gridDim.x = S
    // gridDim.y = B
    __shared__ int wordId;
    __shared__ int tokenId;

    const T rld = T(1.f) / T(ld);
    const int seqPos = blockIdx.y * gridDim.x + blockIdx.x;
    if (threadIdx.x == 0)
    {
        wordId = inputIds[seqPos];
        tokenId = tokenIds[seqPos];
    }
    //if (threadIdx.x == 0 && blockIdx.y == 0 && blockIdx.x <10)
     //   printf("sds %d ,",inputIds[seqPos]);
    __syncthreads();

    // 2. load pos/tok/word embeddings and add them toghether
    // offset into embeddings is given by wordId * hidden_size
    const int poffset = blockIdx.x * ld;
    const int woffset = wordId * ld;
    const int toffset = tokenId * ld;
    // the output offset is given by b * (S*hidden_size) + s * hidden_size
    const int outOffset = seqPos * ld;

    kvp<T> threadData(0, 0);

    for (int it = threadIdx.x; it < ld; it += TPB)
    {
        const T w(wordEmb[woffset + it]);
        const T t(tokEmb[toffset + it]);
        const T p(posEmb[poffset + it]);
        const T val = w + t + p;

        output[outOffset + it] = val;
    //    output[outOffset + it] = wordId;
        const T rldval = rld * val;
        threadData = pairSum(threadData, kvp<T>(rldval, rldval * val));
    }

    // 3. layer norm on the sum
    layerNorm<T, TPB>(threadData, ld, outOffset, beta, gamma, output);
}

template <typename T>
inline int embSkipLayerNorm(hipStream_t stream, int ld, int B, int S, const int* inputIds, const int* token_ids,
    const float* beta, const float* gamma, const float* wordEmb, const float* posEmb, const float* tokEmb, T* output)
{

    constexpr int tpb = 256;
    const dim3 grid(S, B, 1);
    const dim3 block(tpb, 1, 1);
    printf("sds begin hello,\n");
    embLayerNormKernel<T, tpb>
        <<<grid, block, 0, stream>>>(ld, inputIds, token_ids, beta, gamma, wordEmb, posEmb, tokEmb, output);
//    CHECK(hipPeekAtLastError());
    hipError_t cudaerr = hipDeviceSynchronize();
    if (cudaerr != hipSuccess)
        printf("kernel launch failed with error \"%s\".\n",hipGetErrorString(cudaerr));
return 0;
}

// Clip plugin specific constants
namespace
{
static const char* EMB_LAYER_NORM_VERSION{"1"};
static const char* EMB_LAYER_NORM_NAME{"CustomEmbLayerNormPluginDynamic"};
} // namespace

// Static class fields initialization
thread_local PluginFieldCollection EmbLayerNormPluginDynamicCreator::mFC{};
thread_local std::vector<PluginField> EmbLayerNormPluginDynamicCreator::mPluginAttributes;

REGISTER_TENSORRT_PLUGIN(EmbLayerNormPluginDynamicCreator);

EmbLayerNormPluginDynamic::EmbLayerNormPluginDynamic(const std::string& name, const bool outputFp16,
    const Weights& beta, const Weights& gamma, const Weights& wordEmb, const Weights& posEmb, const Weights& tokEmb)
    : mLayerName(name)
    , mLd(beta.count)
    , mGamma(gamma)
    , mBeta(beta)
    , mWordEmb(wordEmb)
    , mPosEmb(posEmb)
    , mTokEmb(tokEmb)
    , mGammaDev(nullptr)
    , mBetaDev(nullptr)
    , mWordEmbDev(nullptr)
    , mTokEmbDev(nullptr)
    , mPosEmbDev(nullptr)
{
    // Assuming Weights.count is the number of elements and not bytes
    assert(beta.count == gamma.count);
    assert(wordEmb.count % mLd == 0);
    assert(posEmb.count % mLd == 0);
    assert(tokEmb.count % mLd == 0);
    mWordVocabSize = wordEmb.count / mLd;
    mPosVocabSize = posEmb.count / mLd;
    mTokVocabSize = tokEmb.count / mLd;
    // We set mS in configure
    mType = outputFp16 ? DataType::kHALF : DataType::kFLOAT;
}

EmbLayerNormPluginDynamic::EmbLayerNormPluginDynamic(const std::string& name, const void* data, size_t length)
    : mLayerName(name)
{
    gLogVerbose << "EMB LN Deser start\n";
    // Deserialize in the same order as serialization
    const char* d = static_cast<const char*>(data);
    const char* a = d;
    DESER(d, mType);
    DESER(d, mLd);
    DESER(d, mS);
    DESER(d, mWordVocabSize);
    DESER(d, mPosVocabSize);
    DESER(d, mTokVocabSize);
    mBetaDev = deserToDev<float>(d, mLd);
    mGammaDev = deserToDev<float>(d, mLd);

    mWordEmbDev = deserToDev<float>(d, mLd * mWordVocabSize);
    mPosEmbDev = deserToDev<float>(d, mLd * mPosVocabSize);
    mTokEmbDev = deserToDev<float>(d, mLd * mTokVocabSize);
    assert(d == (a + length));
    // this signals init not to allocate/copy
    mGamma.count = -1;
    mBeta.count = -1;
    mWordEmb.count = -1;
    mTokEmb.count = -1;
    mPosEmb.count = -1;
    mGamma.values = nullptr;
    mBeta.values = nullptr;
    mWordEmb.values = nullptr;
    mTokEmb.values = nullptr;
    mPosEmb.values = nullptr;

    gLogVerbose << "EMB LN Deser done\n";
}

// IPluginV2DynamicExt Methods
IPluginV2DynamicExt* EmbLayerNormPluginDynamic::clone() const
{
    gLogVerbose << "EMBLN clone start" << std::endl;
    auto ret = new EmbLayerNormPluginDynamic(
        mLayerName, mType == DataType::kHALF, mBeta, mGamma, mWordEmb, mPosEmb, mTokEmb);
    ret->mS = mS;

    ret->mWordEmbDev = mWordEmbDev;
    ret->mPosEmbDev = mPosEmbDev;
    ret->mTokEmbDev = mTokEmbDev;
    ret->mBetaDev = mBetaDev;
    ret->mGammaDev = mGammaDev;
    gLogVerbose << "EMBLN clone done" << std::endl;
    return ret;
}

DimsExprs EmbLayerNormPluginDynamic::getOutputDimensions(int outputIndex, const DimsExprs* inputs, int nbInputs, IExprBuilder& exprBuilder)
{
    // Input should be input ids and token ids and the input mask
    // Output should be the embeddings tensor and mask indices
    assert(nbInputs == 3);

    assert(inputs[0].nbDims == 2); // BxS
    assert(inputs[0].nbDims == inputs[1].nbDims);
    assert(inputs[0].nbDims == inputs[2].nbDims);

    assert(outputIndex == 0 || outputIndex == 1);

    if (outputIndex == 0)
    {
        DimsExprs ret;
        ret.nbDims = 5;
        ret.d[0] = inputs[0].d[0];
        ret.d[1] = inputs[0].d[1];
        ret.d[2] = exprBuilder.constant(mLd);
        ret.d[3] = exprBuilder.constant(1);
        ret.d[4] = exprBuilder.constant(1);
        return ret;
    }

    DimsExprs ret;
    ret.nbDims = 1;
    ret.d[0] = inputs[0].d[BDIM];
    return ret;
}

bool EmbLayerNormPluginDynamic::supportsFormatCombination(int pos, const PluginTensorDesc* inOut, int nbInputs, int nbOutputs)
{
    // 3 inputs of size BxS
    assert(nbInputs == 3);
    assert(nbOutputs == 2);

    const PluginTensorDesc& desc = inOut[pos];
    if (pos == 0)
    {
        return desc.type == DataType::kINT32 && desc.format == TensorFormat::kLINEAR && desc.dims.nbDims == 2;
    }

    const PluginTensorDesc& prev = inOut[pos - 1];
    if (pos == 1 || pos == 2)
    {
        return desc.type == DataType::kINT32 && desc.format == TensorFormat::kLINEAR && desc.dims.nbDims == 2
            && desc.dims.d[BDIM] == prev.dims.d[BDIM] && desc.dims.d[SDIM] == prev.dims.d[SDIM];
    }

    if (pos == 3)
    { // embedded sequence
        return desc.type == mType && desc.format == TensorFormat::kLINEAR && desc.dims.nbDims == 5
            && desc.dims.d[BDIM] == prev.dims.d[BDIM] && desc.dims.d[SDIM] == prev.dims.d[SDIM]
            && desc.dims.d[3] == 1 && desc.dims.d[4] == 1;
    }
    // pos == 4: mask
    return desc.type == DataType::kINT32 && desc.format == TensorFormat::kLINEAR
        && desc.dims.nbDims == 1 && desc.dims.d[BDIM] == prev.dims.d[BDIM];
}

void EmbLayerNormPluginDynamic::configurePlugin(const DynamicPluginTensorDesc* inputs, int nbInputs,
    const DynamicPluginTensorDesc* outputs, int nbOutputs)
{
    // Validate input arguments
    assert(nbOutputs == 2);
    assert(nbInputs == 3);

    assert(inputs[0].desc.dims.nbDims == 2);
    mS = inputs[0].desc.dims.d[SDIM];
    const int B = inputs[0].desc.dims.d[BDIM];
    assert(mS == inputs[1].desc.dims.d[SDIM]);
    assert(B == inputs[1].desc.dims.d[BDIM]);
    assert(mS == inputs[2].desc.dims.d[SDIM]);
    assert(B == inputs[2].desc.dims.d[BDIM]);

    assert(outputs[0].desc.dims.nbDims == 5);
    assert(outputs[0].desc.dims.d[SDIM] == mS);
    assert(outputs[0].desc.dims.d[BDIM] == B);
    assert(outputs[0].desc.dims.d[2] == mLd);
    assert(outputs[0].desc.dims.d[3] == 1);
    assert(outputs[0].desc.dims.d[4] == 1);

    assert(outputs[1].desc.dims.nbDims == 1);
    assert(outputs[1].desc.dims.d[0] == B);

    assert(inputs[0].desc.type== DataType::kINT32);
    assert(inputs[1].desc.type== DataType::kINT32);
    assert(inputs[2].desc.type== DataType::kINT32);
    const DataType out_type = outputs[0].desc.type;
    assert(out_type == DataType::kFLOAT || out_type == DataType::kHALF);
    assert(outputs[1].desc.type == DataType::kINT32);
}

size_t EmbLayerNormPluginDynamic::getWorkspaceSize(const PluginTensorDesc* inputs, int nbInputs,
    const PluginTensorDesc* outputs, int nbOutputs) const
{
    return 0;
}

int EmbLayerNormPluginDynamic::enqueue(const PluginTensorDesc* inputDesc, const PluginTensorDesc* outputDesc,
    const void* const* inputs, void* const* outputs, void* workspace, hipStream_t stream)
{
    const int batchSize = inputDesc->dims.d[BDIM];
    const int S = inputDesc->dims.d[SDIM];
    int status = -1;

    // Our plugin outputs only one tensor
    const int* inputIds = static_cast<const int*>(inputs[0]);
    const int* segmentIds = static_cast<const int*>(inputs[1]);
    const int* inputMask = static_cast<const int*>(inputs[2]);

    if (mType == DataType::kFLOAT)
    {
        float* output = static_cast<float*>(outputs[0]);
        embSkipLayerNorm<float>(stream, mLd, batchSize, S, inputIds, segmentIds, mBetaDev, mGammaDev, mWordEmbDev,
            mPosEmbDev, mTokEmbDev, output);
    }
    else if (mType == DataType::kHALF)
    {
        half* output = static_cast<half*>(outputs[0]);
        embSkipLayerNorm<half>(stream, mLd, batchSize, S, inputIds, segmentIds, mBetaDev, mGammaDev, mWordEmbDev,
            mPosEmbDev, mTokEmbDev, output);
    }
    else
    {
        assert(false);
    }
    int* maskIdx = static_cast<int*>(outputs[1]);
    computeMaskIdx(stream, S, batchSize, inputMask, maskIdx);

    return status;
}

// IPluginV2Ext Methods
DataType EmbLayerNormPluginDynamic::getOutputDataType(int index, const DataType* inputTypes, int nbInputs) const
{

    assert(index == 0 || index == 1);
    if (index == 0)
    {
        assert(mType == DataType::kHALF || mType == DataType::kFLOAT);
        return mType;
    }
    return DataType::kINT32;
}

// IPluginV2 Methods
const char* EmbLayerNormPluginDynamic::getPluginType() const
{
    return EMB_LAYER_NORM_NAME;
}

const char* EmbLayerNormPluginDynamic::getPluginVersion() const
{
    return EMB_LAYER_NORM_VERSION;
}

int EmbLayerNormPluginDynamic::getNbOutputs() const
{
    return 2;
}

int EmbLayerNormPluginDynamic::initialize()
{
    if (mGamma.values)
    {
        CHECK(hipMalloc(&mGammaDev, sizeof(float) * mGamma.count));
        CHECK(hipMemcpy(mGammaDev, mGamma.values, sizeof(float) * mGamma.count, hipMemcpyHostToDevice));
    }
    if (mBeta.values)
    {
        CHECK(hipMalloc(&mBetaDev, sizeof(float) * mBeta.count));
        CHECK(hipMemcpy(mBetaDev, mBeta.values, sizeof(float) * mBeta.count, hipMemcpyHostToDevice));
    }

    if (mWordEmb.values)
    {
        CHECK(hipMalloc(&mWordEmbDev, sizeof(float) * mWordEmb.count));
        CHECK(hipMemcpy(mWordEmbDev, mWordEmb.values, sizeof(float) * mWordEmb.count, hipMemcpyHostToDevice));
    }
    if (mTokEmb.values)
    {
        CHECK(hipMalloc(&mTokEmbDev, sizeof(float) * mTokEmb.count));
        CHECK(hipMemcpy(mTokEmbDev, mTokEmb.values, sizeof(float) * mTokEmb.count, hipMemcpyHostToDevice));
    }

    if (mPosEmb.values)
    {
        CHECK(hipMalloc(&mPosEmbDev, sizeof(float) * mPosEmb.count));
        CHECK(hipMemcpy(mPosEmbDev, mPosEmb.values, sizeof(float) * mPosEmb.count, hipMemcpyHostToDevice));
    }
    return 0;
}

void EmbLayerNormPluginDynamic::terminate()
{
    gLogVerbose << "EMBLN terminate start" << std::endl;
    CHECK(hipFree(mGammaDev));
    CHECK(hipFree(mBetaDev));
    CHECK(hipFree(mWordEmbDev));
    CHECK(hipFree(mTokEmbDev));
    CHECK(hipFree(mPosEmbDev));
    gLogVerbose << "EMBLN terminate done" << std::endl;
}

size_t EmbLayerNormPluginDynamic::getSerializationSize() const
{
    return 2 * sizeof(float) * mLd             // beta + gamma
        + sizeof(mType) + sizeof(mLd) * 5      //mLd, mS, m*VocabSize
        + sizeof(float) * mLd * mWordVocabSize // word emb
        + sizeof(float) * mLd * mPosVocabSize  // pos emb
        + sizeof(float) * mLd * mTokVocabSize  // tok emb
        ;
}

void EmbLayerNormPluginDynamic::serialize(void* buffer) const
{
    char* d = static_cast<char*>(buffer);
    const char* a = d;
    writeToBuffer(d, mType);
    writeToBuffer(d, mLd);
    writeToBuffer(d, mS);
    writeToBuffer(d, mWordVocabSize);
    writeToBuffer(d, mPosVocabSize);
    writeToBuffer(d, mTokVocabSize);
    serFromDev(d, mBetaDev, mLd);
    serFromDev(d, mGammaDev, mLd);
    serFromDev(d, mWordEmbDev, mLd * mWordVocabSize);
    serFromDev(d, mPosEmbDev, mLd * mPosVocabSize);
    serFromDev(d, mTokEmbDev, mLd * mTokVocabSize);

    assert(d == a + getSerializationSize());
}

void EmbLayerNormPluginDynamic::destroy()
{
    gLogVerbose << "EMBLN destroy start" << std::endl;
    // This gets called when the network containing plugin is destroyed
    delete this;
    gLogVerbose << "EMBLN destroy start" << std::endl;
}

void EmbLayerNormPluginDynamic::setPluginNamespace(const char* libNamespace)
{
    mNamespace = libNamespace;
}

const char* EmbLayerNormPluginDynamic::getPluginNamespace() const
{
    return mNamespace.c_str();
}

///////////////////////

EmbLayerNormPluginDynamicCreator::EmbLayerNormPluginDynamicCreator()
{
    mFC.nbFields = mPluginAttributes.size();
    mFC.fields = mPluginAttributes.data();
}

const char* EmbLayerNormPluginDynamicCreator::getPluginName() const
{
    return EMB_LAYER_NORM_NAME;
}

const char* EmbLayerNormPluginDynamicCreator::getPluginVersion() const
{
    return EMB_LAYER_NORM_VERSION;
}

const PluginFieldCollection* EmbLayerNormPluginDynamicCreator::getFieldNames()
{
    return &mFC;
}

IPluginV2* EmbLayerNormPluginDynamicCreator::createPlugin(const char* name, const PluginFieldCollection* fc)
{
    gLogVerbose << "Creating EmbLayerNormPluginDynamic...\n";

    bool output_fp16 = true;
    Weights beta;
    Weights gamma;
    Weights word_emb;
    Weights pos_emb;
    Weights tok_emb;
    for (int i = 0; i < fc->nbFields; i++)
    {
        std::string field_name(fc->fields[i].name);
        if (field_name.compare("bert_embeddings_layernorm_beta") == 0)
        {
            gLogVerbose << "Building bert_embeddings_layernorm_beta...\n";
            beta.values = fc->fields[i].data;
            beta.count = fc->fields[i].length;
            beta.type = static_cast<DataType>(fc->fields[i].type);
        }

        if (field_name.compare("bert_embeddings_layernorm_gamma") == 0)
        {
            gLogVerbose << "Building bert_embeddings_layernorm_gamma...\n";
            gamma.values = fc->fields[i].data;
            gamma.count = fc->fields[i].length;
            gamma.type = static_cast<DataType>(fc->fields[i].type);
        }

        if (field_name.compare("bert_embeddings_word_embeddings") == 0)
        {
            gLogVerbose << "Building bert_embeddings_word_embeddings...\n";
            word_emb.values = fc->fields[i].data;
            word_emb.count = fc->fields[i].length;
            word_emb.type = static_cast<DataType>(fc->fields[i].type);
        }

        if (field_name.compare("bert_embeddings_token_type_embeddings") == 0)
        {
            gLogVerbose << "Building bert_embeddings_token_type_embeddings...\n";
            tok_emb.values = fc->fields[i].data;
            tok_emb.count = fc->fields[i].length;
            tok_emb.type = static_cast<DataType>(fc->fields[i].type);
        }

        if (field_name.compare("bert_embeddings_position_embeddings") == 0)
        {
            gLogVerbose << "Building bert_embeddings_position_embeddings...\n";
            pos_emb.values = fc->fields[i].data;
            pos_emb.count = fc->fields[i].length;
            pos_emb.type = static_cast<DataType>(fc->fields[i].type);
        }
    }

    gLogVerbose << "Building the Plugin...\n";
    EmbLayerNormPluginDynamic* p
        = new EmbLayerNormPluginDynamic(name, output_fp16, beta, gamma, word_emb, pos_emb, tok_emb);
    return p;
}

IPluginV2* EmbLayerNormPluginDynamicCreator::deserializePlugin(
    const char* name, const void* serialData, size_t serialLength)
{
    // This object will be deleted when the network is destroyed, which will
    // call EmbLayerNormPluginDynamic::destroy()
    return new EmbLayerNormPluginDynamic(name, serialData, serialLength);
}

void EmbLayerNormPluginDynamicCreator::setPluginNamespace(const char* libNamespace)
{
    mNamespace = libNamespace;
}

const char* EmbLayerNormPluginDynamicCreator::getPluginNamespace() const
{
    return mNamespace.c_str();
}
}
}
